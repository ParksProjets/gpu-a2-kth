
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>


// Constant values.
int NUM_PARTICLES  = 10000;
int NUM_ITERATIONS = 1000;
int BLOCK_SIZE     = 32;

// Whether to use `cudaMallocHost` or not.
#define USE_CUDA_MALLOC 1


// Data of a single particule.
struct Particle {
    float3 position;
    float3 velocity;
};


// Generate a random number between in range [a, b].
#define RAND_FLOAT(a,b) (a + (float)rand() / RAND_MAX * (b-a))

// Check if the given command has returned an error.
#define CUDA_CHECK(cmd) if ((cmd) != hipSuccess) { \
    printf("ERROR: cuda error at line %d\n", __LINE__); abort(); }


// Initialize the array of particules.
Particle *CreateParticuleArray()
{
    srand(42);

#if USE_CUDA_MALLOC
    Particle *array;
    hipHostMalloc(&array, sizeof(Particle) * NUM_PARTICLES, hipHostMallocDefault);
#else
    Particle *array = (Particle *)malloc(sizeof(Particle) * NUM_PARTICLES);
#endif

    for (int index = 0; index < NUM_PARTICLES; index++) {
        array[index].velocity.x = RAND_FLOAT(1, 10);
        array[index].velocity.y = RAND_FLOAT(1, 20);
        array[index].velocity.z = RAND_FLOAT(5, 30);

        array[index].position.x = RAND_FLOAT(-100, 100);
        array[index].position.y = RAND_FLOAT(-100, 100);
        array[index].position.z = RAND_FLOAT(-100, 100);
    }

    return array;
}


// Update a particule by one single step.
__device__ void UpdateParticule(Particle &particule, const float3 &dvel)
{
    particule.velocity.x += dvel.x;
    particule.velocity.y += dvel.y;
    particule.velocity.z += dvel.z;

    particule.position.x += particule.velocity.x;
    particule.position.y += particule.velocity.y;
    particule.position.z += particule.velocity.z;
}


// GPU kernel for updating a particule by one single step.
__global__ void GpuUpdate(Particle *particules, int NUM_PARTICLES, float3 dvel)
{
    auto index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < NUM_PARTICLES)
        UpdateParticule(particules[index], dvel);
}


// Make all iterations on GPU.
void GpuInterations(Particle *array, Particle *gpuarray, float3 dvel)
{
    int num_blocks = (NUM_PARTICLES + BLOCK_SIZE - 1) / BLOCK_SIZE;

    for (int i = 0; i < NUM_ITERATIONS; i++) {
        CUDA_CHECK(hipMemcpy(gpuarray, array, sizeof(Particle) * NUM_PARTICLES, hipMemcpyHostToDevice));

        GpuUpdate<<<num_blocks, BLOCK_SIZE>>>(gpuarray, NUM_PARTICLES, dvel);
        hipDeviceSynchronize();  // Make sure the particules were updated.

        CUDA_CHECK(hipMemcpy(array, gpuarray,  sizeof(Particle) * NUM_PARTICLES, hipMemcpyDeviceToHost));
    }
}


// Entry point of this program.
int main(int argc, const char **argv)
{
    // When the program is ran with -h, show usage.
    if (argc == 2 && !strcmp(argv[1], "-h")) {
        printf("Usage: ./exercise_2a [num particules] [num iterations] [block size]\n");
        exit(0);
    }

    // Read number of particules, number of iterations and block size.
    if (argc >= 2) NUM_PARTICLES  = atoi(argv[1]);
    if (argc >= 3) NUM_ITERATIONS = atoi(argv[2]);
    if (argc >= 4) BLOCK_SIZE     = atoi(argv[3]);

    // Velocity increment on each step.
    float3 dvel = make_float3(-1.f, 3.45f, 7.3f);


    // Run iterations on GPU.
    Particle *gpuarray;
    Particle *array = CreateParticuleArray();
    CUDA_CHECK(hipMalloc(&gpuarray, sizeof(Particle) * NUM_PARTICLES));

    printf("\nStarting GPU test ...\n");
    auto start = std::chrono::system_clock::now();

    GpuInterations(array, gpuarray, dvel);

    auto end = std::chrono::system_clock::now();
    int ms = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printf("GPU time: %d ms\n\n", ms);
}
